#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>
#include "cuPrintf.cu" /* For debugging */
#include "cuPrintf.cuh" /* For debugging */

#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif


__global__ void cudasort(float *data, float *temp, int num_of_elements, 
        int subarr_size) {

    unsigned long index;          /* index when copying between temp and data*/
    unsigned long global_index;   /* index of thread */
    unsigned long left_lower;     /* start of left subarray */
    unsigned long mid;            /* end of left subarray */
    unsigned long right_upper;    /* end of right subarray */
    unsigned long right_lower;    /* start orf right subarray (aka mid + 1) */

    /* Indexing in CUDA is hard. Due to the grid-like nature, getting the i-th
     * thread means calculating the amount of threads/row times the row the i-th
     * thread is on, add this to the amount of threads in the row until you 
     * reach the block the i-th thread is in, then adding this to the amount of
     * threads until you reach the i-th thread */
    global_index = (gridDim.x * blockDim.x * blockIdx.y) + 
                   (blockDim.x * blockIdx.x) + threadIdx.x;

    /* i-th thread works on i-th and i-th + 1 subarrays, so don't do anything 
     * when i-th subarray is more than total subarrays */
    left_lower = global_index * subarr_size;
    if(left_lower >= num_of_elements) {
        return;
    }

    /* Mergesort works on A[p..q] and A[q+1..r], so calculate p == left_lower,
     * q = mid, q+1 = mid + 1 = right_lower, and r = right_upper. Last subarray
     * may end up being shorter due to being the remaining amount of the data.*/
    right_upper = (left_lower + subarr_size - 1) >= (num_of_elements - 1) ? 
        (num_of_elements - 1) : (left_lower + subarr_size - 1) ;
    mid = (left_lower + right_upper) / 2;
    right_lower = mid + 1;
    index = left_lower;

    /* Begin merging. While we still have elements to merge with... */
    while(left_lower <= mid || right_lower <= right_upper) {

        /* if left subarray is parsed, grab from the right subarray */
        if(left_lower > mid) {
            temp[index++] = data[right_lower++];
        }

        /* if right subarray is parsed, grab from the left subarray */
        else if(right_lower > right_upper) {
            temp[index++] = data[left_lower++];
        }

        /* if next smallest element is in right subarray, grab it */
        else if(data[right_lower] < data[left_lower]) {
            temp[index++] = data[right_lower++];
        }

        /* if next smallest element is in left subarray, grab it */
        else if(data[left_lower] <= data[right_lower]) {
            temp[index++] = data[left_lower++];
        }
    }

    /* Copy the semi-sorted temp content back to the original data set */
    for(index = global_index * subarr_size; index <= right_upper; index++) {
        data[index] = temp[index];
    }
}


int cuda_sort(int num_of_elements, float *data)
{
    float *cuda_data;  /* gpu-side storage for data parameter */
    float *cuda_temp;  /* gpu-side storage for temp work buffer */
    float *temp;       /* temporary storage that acts like a workspace */
    int subarr_size;   /* amount of elements in subarrays when merging */

    unsigned long size_in_bytes = num_of_elements * sizeof(float);

    /* Create buffers for initial data and temp buffer */
    temp = (float *) malloc(size_in_bytes);
    memset(temp, 0, size_in_bytes);
    hipMalloc((void **) &cuda_data, size_in_bytes);
    hipMalloc((void **) &cuda_temp, size_in_bytes);

    /* Merge sort is recursive, but OpenCL doesn't allow recursion (janky)
     * so instead mergesort is then iterative. Each loop iteration is the 
     * next up recursion level starting with the leaf nodes of the recursion
     * tree. NOTE this implementation requires the data to be a power of 2 and
     * _will not work_ otherwise. If it does, it's coincidental and I didn't
     * make it happen :) */
    for(subarr_size = 2; subarr_size <= num_of_elements; 
            subarr_size = subarr_size * 2) {

        /* Final data size is 16777216 so problem space needs to hold this. 256
         * x 256 x 512 (or more precisely a 256 x 256 grid each with 512 x 1 
         * blocks) should suffice. */
        dim3 dimGrid(256, 256);

        /* Copy stuff to cuda buffers */
        hipMemcpy(cuda_data, data, size_in_bytes, hipMemcpyHostToDevice);
        hipMemcpy(cuda_temp, temp, size_in_bytes, hipMemcpyHostToDevice);

        /* Execute kernel, 2D grid, (256, 256), with 1D blocks, (512, 1) */
        cudasort<<<dimGrid, 512>>>(cuda_data, cuda_temp, num_of_elements, 
                subarr_size);

        /* Read data from GPU (either partially or fully sorted) */
        hipMemcpy(data, cuda_data, size_in_bytes, hipMemcpyDeviceToHost);
    }

    /* Clean up */
    free(temp);
    hipFree(cuda_data);
    hipFree(cuda_temp);
    return 0;
}

#ifdef __cplusplus
}
#endif

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>
#include "cuPrintf.cu" /* For debugging */
#include "cuPrintf.cuh" /* For debugging */

#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif


__global__ void cudasort(float *data, float *temp, int num_of_elements, 
        int subarr_size) {

    unsigned long index;          /* index when copying between temp and data*/
    unsigned long global_index;   /* index of thread */
    unsigned long left_lower;     /* start of left subarray */
    unsigned long mid;            /* end of left subarray */
    unsigned long right_upper;    /* end of right subarray */
    unsigned long right_lower;    /* start orf right subarray (aka mid + 1) */
    unsigned long temp_index;     /* Index into semi-sorted interim result buffer */

    /* ith work item works on ith and ith + 1 subarrays,
     * so don't do anything when ith subarray is more than
     * total subarrays */
    global_index = (blockIdx.x * gridDim.x) + (blockIdx.y * gridDim.y) + threadIdx.x;
    left_lower = global_index * subarr_size;
    if(left_lower >= num_of_elements) {
        return;
    }

    /* Mergesort works on A[p..q] and A[q+1..r], so calculate p == left_lower,
     * q = mid, q+1 = mid + 1 = right_lower, and r = right_upper
     */
    right_upper = (left_lower + subarr_size - 1) >= (num_of_elements - 1) ? (num_of_elements - 1) : (left_lower + subarr_size - 1) ;
    mid = (left_lower + right_upper) / 2;
    right_lower = mid + 1;

    temp_index = left_lower;
    while(left_lower <= mid || right_lower <= right_upper) {
        if(left_lower > mid) {
            temp[temp_index++] = data[right_lower++];
        }
        else if(right_lower > right_upper) {
            temp[temp_index++] = data[left_lower++];
        }
        else if(data[right_lower] < data[left_lower]) {
            temp[temp_index++] = data[right_lower++];
        }
        else if(data[left_lower] <= data[right_lower]) {
            temp[temp_index++] = data[left_lower++];
        }
    }

    __syncthreads();
    /* Copy the semi-sorted temp content back to the original data set */
    for(index = global_index * subarr_size; index <= right_upper; index++) {
        data[index] = temp[index];
    }

}


int cuda_sort(int num_of_elements, float *data)
{
    float *cuda_data;
    float *cuda_temp;

    float *temp;
    int subarr_size;


    unsigned long size_in_bytes = num_of_elements * sizeof(float);

    /* Create buffers for initial data and temp buffer */
    hipMalloc((void **) &cuda_data, size_in_bytes);
    hipMalloc((void **) &cuda_temp, size_in_bytes);
    temp = (float *) malloc(size_in_bytes);
    memset(temp, 0, size_in_bytes);


    cudaPrintfInit ();  /* For debugging */


    /* Merge sort is recursive, but OpenCL doesn't allow recursion (janky)
     * so instead mergesort is then iterative. Each loop iteration is the 
     * next up recursion level starting with the leaf nodes of the recursion
     * tree. */
    //int i;
    for(subarr_size = 2; subarr_size <= num_of_elements; 
            subarr_size = subarr_size * 2) {

        dim3 dimGrid(256, 256);
        //dim3 dimBlock(512, 1);
        /* Copy stuff to cuda buffers */
        hipMemcpy(cuda_data, data, size_in_bytes, hipMemcpyHostToDevice);
        hipMemcpy(cuda_temp, temp, size_in_bytes, hipMemcpyHostToDevice);

        /* Execute kernel */
        //cudasort<<<1, num_of_elements>>>(cuda_data, cuda_temp, 
        //        num_of_elements, subarr_size);
        cudasort<<<dimGrid, 512>>>(cuda_data, 
                cuda_temp, num_of_elements, subarr_size);
        hipDeviceSynchronize();

        /* Read data from GPU (either partially or fully sorted) */
        hipMemcpy(data, cuda_data, size_in_bytes, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }

    /* Clean up */
        cudaPrintfDisplay (stdout, true);  /* For debugging */
    cudaPrintfEnd ();  /* For debugging */ 

    free(temp);
    hipFree(cuda_data);
    hipFree(cuda_temp);
    return 0;
}

#ifdef __cplusplus
}
#endif

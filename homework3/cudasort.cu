
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>

#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif

__global__ void cudasort(float *data, float *temp, int num_of_elements, 
        int subarr_size) {
}

int cuda_sort(int num_of_elements, float *data)
{
    float *cuda_data;
    float *cuda_temp;

    float *temp;
    int subarr_size;
    unsigned long size_in_bytes = num_of_elements * sizeof(float);

    /* Create buffers for initial data and temp buffer */
    hipMalloc((void **) &cuda_data, size_in_bytes);
    hipMalloc((void **) &cuda_temp, size_in_bytes);
    temp = (float *) malloc(size_in_bytes);
    memset(temp, 0, size_in_bytes);


    /* Merge sort is recursive, but OpenCL doesn't allow recursion (janky)
     * so instead mergesort is then iterative. Each loop iteration is the 
     * next up recursion level starting with the leaf nodes of the recursion
     * tree. */
    for(subarr_size = 2; subarr_size <= num_of_elements; 
            subarr_size = subarr_size * 2) {
        printf("%d Hidy-ho kids I'm mister Hanky\n", subarr_size); 

        /* Copy stuff to cuda buffers */
        hipMemcpy(cuda_data, data, size_in_bytes, hipMemcpyHostToDevice);
        hipMemcpy(cuda_temp, temp, size_in_bytes, hipMemcpyHostToDevice);

        /* Execute kernel */
        cudasort<<<num_of_elements, subarr_size>>>(cuda_data, cuda_temp, 
                num_of_elements, subarr_size);

        /* Read data from GPU (either partially or fully sorted) */
        hipMemcpy(data, cuda_data, size_in_bytes, hipMemcpyHostToDevice);
    }

    /* Clean up */
    free(temp);
    hipFree(cuda_data);
    hipFree(cuda_temp);
    return 0;
}

#ifdef __cplusplus
}
#endif
